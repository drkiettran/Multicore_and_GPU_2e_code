/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2019
 License       : Released under the GNU GPL 3.0
 Description   : 
 To build use  : nvcc graphCaptureExample.cu -o graphCaptureExample
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void A()
{
  printf ("A\n");
}
__global__ void B()
{
  printf ("B\n");
}
__global__ void C()
{
  printf ("C\n");
}
__global__ void D()
{
  printf ("D\n");
}
__global__ void E()
{
  printf ("E\n");
}
__global__ void F()
{
  printf ("F\n");
}

int main ()
{
  hipStream_t str1, str2, execStream;
  
  hipStreamCreate(&str1);
  hipStreamCreate(&str2);
  hipStreamCreate(&execStream);
  hipGraph_t gr;
  hipEvent_t eventB2C, eventC2D, startEvent, finishEvent;
  hipEventCreate(&eventB2C);
  hipEventCreate(&eventC2D);
  hipEventCreate(&startEvent);
  hipEventCreate(&finishEvent);
  
  // origin stream 
  hipStreamBeginCapture(str1, hipStreamCaptureModeGlobal);
  hipEventRecord(startEvent, str1); 
  A<<<1,1,0,str1>>>();
  hipStreamWaitEvent(str1, eventB2C,0);
  C<<<1,1,0,str1>>>();
  hipEventRecord(eventC2D, str1);
  E<<<1,1,0,str1>>>();

  // second captured stream
  hipStreamWaitEvent(str2, startEvent,0);
  B<<<1,1,0,str2>>>();
  hipEventRecord(eventB2C, str2);
  hipStreamWaitEvent(str2, eventC2D,0);
   D<<<1,1,0,str2>>>();
   F<<<1,1,0,str2>>>();
  hipEventRecord(finishEvent, str2);
   
  hipStreamWaitEvent(str1, finishEvent,0);
  hipStreamEndCapture(str1, &gr);
  
  //***********************************************************
  // Instantiation phase
  hipGraphExec_t instance;
  hipGraphInstantiate (&instance, gr, NULL, NULL, 0);
  
  //***********************************************************
  // Execution phase
  hipGraphLaunch (instance, execStream);  
  hipStreamSynchronize(execStream);
  
  hipEventDestroy(eventB2C);
  hipEventDestroy(eventC2D);
  hipStreamDestroy(str1);
  hipStreamDestroy(str2);
  hipDeviceReset();
  return 1;
}
