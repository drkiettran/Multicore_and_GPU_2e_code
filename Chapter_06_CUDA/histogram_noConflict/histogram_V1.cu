#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.1
 Last modified : November 2019
 License       : Released under the GNU GPL 3.0
 Description   : Maximum number of bins are used
                 warpSize is assumed to be fixed to 32
 To build use  : make
 ============================================================================
 */

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <memory>
#include <hip/hip_runtime.h>
#include "../common/pgm.h"

using namespace std;

const int BINS = 256;
const int BINS4ALL = BINS * 32;

//*****************************************************************
void CPU_histogram (unsigned char *in, int N, int *h, int bins)
{
  int i;
  // initialize histogram counts
  for (i = 0; i < bins; i++)
    h[i] = 0;

  // accummulate counts
  for (i = 0; i < N; i++)
    h[in[i]]++;
}

//*****************************************************************
__global__ void GPU_histogram_V1 (int *in, int N, int *h)
{
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;
  int locID = threadIdx.x;
  int GRIDSIZE = gridDim.x * blockDim.x;
  __shared__ int localH[BINS4ALL];
  int bankID = locID & 0x1F;    // Optimized version of locID % warpSize;
  int i;

  // initialize the local, shared-memory bins
  for (i = locID; i < BINS4ALL; i += blockDim.x)
    localH[i] = 0;

  // wait for all warps to complete the previous step
  __syncthreads ();

  //start processing the image data
  for (i = gloID; i < N; i += GRIDSIZE)
    {
      int temp = in[i];
      int v = temp & 0xFF;
      localH[bankID + (v << 5)]++;      // Optimized version of localH[bankID + v * warpSize]++
      v = (temp >> 8) & 0xFF;
      localH[bankID + (v << 5)]++;
      v = (temp >> 16) & 0xFF;
      localH[bankID + (v << 5)]++;
      v = (temp >> 24) & 0xFF;
      localH[bankID + (v << 5)]++;
    }

  // wait for all warps to complete the local calculations, before updating the global counts
  __syncthreads ();

  // use atomic operations to add the local findings to the global memory bins 
  for (i = locID; i < BINS4ALL; i += blockDim.x)
    atomicAdd (h + (i >> 5), localH[i]);        // Optimized version of atomicAdd (h + (i/warpSize), localH[i]);
}

//*****************************************************************
int main (int argc, char **argv)
{

  PGMImage inImg (argv[1]);

  unique_ptr<int[]> h_hist, cpu_hist;
  int *h_in, *d_in;
  int *d_hist;
  int i, N, bins;

  h_in = (int *) inImg.pixels;
  N = ceil ((inImg.x_dim * inImg.y_dim) / 4.0);

  bins = inImg.num_colors + 1;
  h_hist = make_unique<int[]>(bins);
  cpu_hist = make_unique<int[]>(bins);

  CPU_histogram (inImg.pixels, inImg.x_dim * inImg.y_dim, cpu_hist.get(), bins);

  hipMalloc ((void **) &d_in, sizeof (int) * N);
  hipMalloc ((void **) &d_hist, sizeof (int) * bins);
  hipMemcpy (d_in, h_in, sizeof (int) * N, hipMemcpyHostToDevice);
  hipMemset (d_hist, 0, bins * sizeof (int));

  hipDeviceProp_t pr;
  hipGetDeviceProperties (&pr, 0);     // replace 0 with appropriate ID in case of a multi-GPU system
  int SM = pr.multiProcessorCount;
  int blockSize = 32;
  int gridSize = min( SM, (int)ceil (1.0 * N / blockSize));
  printf ("Grid : %i\n", gridSize);
  
// timing related definitions  
  hipStream_t str;
  hipEvent_t startT, endT;
  float duration;

// initialize two events
  hipStreamCreate (&str);
  hipEventCreate (&startT);
  hipEventCreate (&endT);

  hipEventRecord (startT, str);
  GPU_histogram_V1 <<< gridSize, blockSize, 0, str >>> (d_in, N, d_hist);
  hipEventRecord (endT, str);

// wait for endT event to take place
  hipEventSynchronize (endT);

  hipMemcpy (h_hist.get(), d_hist, sizeof (int) * bins, hipMemcpyDeviceToHost);

//  for (i = 0; i < BINS; i++)
//    printf ("%i %i %i\n", i, cpu_hist[i], h_hist[i]);

  for (i = 0; i < BINS; i++)
    if (cpu_hist[i] != h_hist[i])
      printf ("Calculation mismatch (static) at : %i\n", i);

// calculate elapsed time
  hipEventElapsedTime (&duration, startT, endT);
  printf ("Kernel executed for %f ms\n", duration);

// clean-up allocated objects and reset device
  hipStreamDestroy (str);
  hipEventDestroy (startT);
  hipEventDestroy (endT);

  hipFree ((void *) d_in);
  hipFree ((void *) d_hist);
  hipDeviceReset ();

  return 0;
}
