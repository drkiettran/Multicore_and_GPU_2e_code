/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2019
 License       : Released under the GNU GPL 3.0
 Description   : 
 To build use  : nvcc odd_warpPrim.cu -o odd_warpPrim
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <memory>

#define MAXVALUE 10000

using namespace std;

//------------------------------------
void numberGen (int N, int max, int *store)
{
  int i;
  srand (time (0));
  for (i = 0; i < N; i++)
    store[i] = rand () % max;
}

//------------------------------------

__global__ void countOdds (int *d, int N, int *odds)
{
  extern __shared__ int count[];

  int myID = blockIdx.x * blockDim.x + threadIdx.x;
  int totalThr = blockDim.x * gridDim.x;
  int localID = threadIdx.x;
  count[localID] = 0;
  for(int i=myID; i < N; i+= totalThr)
     count[localID] += (d[i] % 2);
  __syncthreads ();

  // reduction phase 1 : sum up the warp results
  unsigned mask = 0xffffffff;
  int val = count[localID];
//   printf("# %i %i\n", localID %32, val);
  for(int offset=16;offset>0;offset/=2)
    {
     val += __shfl_down_sync(mask, val, offset);
//      printf("%i %i %i\n", offset, localID %32, val);
    } 
  if(localID % 32 == 0) // lane 0 in all warps
     count[localID]=val;
     
   __syncthreads ();
  
  // reduction phase 2: sum up the warp leaders results
  int step = 32;
  int otherIdx = localID | step;  
  while ((otherIdx < blockDim.x) && ((localID & step) == 0) )
    {
      count[localID] += count[otherIdx];
      step <<= 1;
      otherIdx = localID | step;  
      __syncthreads ();
    }
    
  // add to global counter
  if (localID == 0)
    atomicAdd (odds, count[0]);
}

//------------------------------------
int sharedSize (int b)
{
  return b * sizeof (int);
}

//------------------------------------

int main (int argc, char **argv)
{
  int N = atoi (argv[1]);

  unique_ptr<int[]> ha; // host (h*) and
  int *da;              // device (d*) pointers
  int *dres;
  int hres; 
  
  ha = make_unique<int[]>(N);

  hipMalloc ((void **) &da, sizeof (int) * N);
  hipMalloc ((void **) &dres, sizeof (int) * 1);

  numberGen (N, MAXVALUE, ha.get());

  hipMemcpy (da, ha.get(), sizeof (int) * N, hipMemcpyHostToDevice);
  hipMemset (dres, 0, sizeof (int));

  hipDeviceProp_t pr;
  hipGetDeviceProperties (&pr, 0);     // replace 0 with appropriate ID in case of a multi-GPU system
  int SM = pr.multiProcessorCount;
  
  int blockSize = 256;
  int blockPerSM, gridSize;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockPerSM, (void *) countOdds, blockSize, sharedSize(blockSize));
   
  gridSize = min( (int)ceil (1.0 * N / blockSize), blockPerSM * SM);

  printf ("Grid : %i    Block : %i  Suggested: %i\n", gridSize, blockSize, blockPerSM * SM);
  countOdds <<< gridSize, blockSize, blockSize * sizeof (int) >>> (da, N, dres);

  hipMemcpy (&hres, dres, sizeof (int), hipMemcpyDeviceToHost);

  // correctness check
  int oc = 0;
  for (int i = 0; i < N; i++)
    if (ha[i] % 2)
      oc++;

  printf ("%i %i\n", hres, oc);

  hipFree ((void *) da);
  hipFree ((void *) dres);
  hipDeviceReset ();

  return 0;
}
